
#include <hip/hip_runtime.h>

#define N 256

__global__ void add(int *pa, int *pb, int *pc)
{
    int tid = blockIdx.x;
    if (tid < N) 
        pc[tid] = pa[tid] + pb[tid];
}

int main()
{
    int M = 1 << 16; // 

    int ha[M], hb[M], hc[M];
    int *da, *db, *dc;

    ha[0] = hb[0] = hc[0] = 1;
    for (int i = 1; i < M; i++) {
        ha[i] = (2*ha[i-1] + 7) % 753;
        hb[i] = (3*hb[i-1] + 17) % 753;
        hc[i] = (5*hc[i-1] + 47) % 753;
    }

    hipMalloc((void**) &da, N*sizeof(int));
    hipMalloc((void**) &db, N*sizeof(int));
    hipMalloc((void**) &dc, N*sizeof(int));

    for (int i = 0; i < M/N; i++) {
        hipMemcpy(da, ha+i*N, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(db, hb+i*N, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dc, hc+i*N, N*sizeof(int), hipMemcpyHostToDevice);

        add<<<N, 1>>>(da, db, dc);

        hipMemcpy(hc+i*N, dc, N*sizeof(int), hipMemcpyDeviceToHost);
        // for (int j = 0; j < N; )
    }
    // cudaMemcpy();
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}
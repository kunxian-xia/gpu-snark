#include "hip/hip_runtime.h"

#define N 256

__global__ void add(int *pa, int *pb, int *pc)
{
    int tid = blockIdx.x;
    if (tid < N) 
        c[tid] = a[tid] + b[tid];
}

int main()
{
    int M = 1 << 16; // 

    int ha[M], hb[M], hc[M];
    int *da, *db, *dc;

    ha[0] = hb[0] = hc[0] = 1;
    for (int i = 1; i < M; i++) {
        ha[i] = (2*ha[i-1] + 7) % 753;
        hb[i] = (3*hb[i-1] + 17) % 753;
        hc[i] = (5*hc[i-1] + 47) % 753;
    }

    hipMalloc((void**) &da, N*sizeof(int));
    hipMalloc((void**) &db, N*sizeof(int));
    hipMalloc((void**) &dc, N*sizeof(int));

    for (int i = 0; i < M/N; i++) {
        hipMemcpy(da, a+i*N, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(db, b+i*N, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dc, c+i*N, N*sizeof(int), hipMemcpyHostToDevice);

        add<<<N, 1>>>(da, db, dc);

        hipMemcpy(c+i*N, dc, N*sizeof(int), hipMemcpyDeviceToHost);
        // for (int j = 0; j < N; )
    }
    // hipMemcpy();
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}